#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <MatKernel.hpp>
#include <MurmurHash.hpp>

#if __CUDA_ARCH__ >= 300
#define MAXXGRID 2147483647
#else
#define MAXXGRID 65535
#endif

void setsizes(int N, dim3 *gridp, int *nthreadsp);

__forceinline__ __device__ int solve1(int j) {
  float v = sqrtf((float)j);
#pragma unroll
  for (int k = 0; k < 5; k++) {
    v = v - (v*(v+1)-2*j)/(2*v+1);   // Newton iterations to find first index. 
  }
  return (int)(v+2e-5f);   
}

__forceinline__ __device__ void solvex(int n, int v, int &i, int &j) {
  int n1 = ((n >> 1) << 1) + 1;
  int n2 = (n + 1) >> 1;
  int even = (n1 != n);
  j = v / n1;
  i = v - n1 * j;
  if (j > i - even) {
    i = n1 - i - 1;
    j = n2 + n2 - j + 1;
  } else {
    i = i - even;
  }
}

// Feature hashing multiply and multiply-transpose.
// This one enumerates, hashes and multiplies all pairs of features.
//
// NOTE: The single-matrix version (hashmult) uses a fast lookup recurrence which is only valid up to 3000 base features per column (approx 4.5 million pairs)

// Given dense A and sparse B, for each column of B, enumerate all pairs of features, hash to a single feature index, and multiply by A into C

__global__ void __hashmult(int nrows, int nfeats, int bncols, int brows1, int brows2, float *A, float *Bdata, int *Bir, int *Bjc, float *C, int transpose) {
  bool doit = false;
  int istart = ((long long)blockIdx.x) * bncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * bncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i];                                     // Range of nz rows in this column
    int jend = Bjc[i+1];
    int nr = jend - jstart;                                  // Number of nz rows
    int todo = nr * (nr + 1) / 2;                            // Number of pairs to process (including k,k pairs)
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = solve1(j);                                    // Compute the first and second indices
      int j2 = j - j1*(j1+1)/2; 
      //      int j1, j2;
      //      solvex(todo, j, j1, j2);
      float f1 = Bdata[jstart + j1];                         // Get the two features
      float f2 = Bdata[jstart + j2];
      int r1 = Bir[jstart + j1];                             // And their row indices
      int r2 = Bir[jstart + j2];
      long long rank = r1 + 1;
      float prod = f1;
      if (j1 == j2) {
        doit = (rank < brows1);
      } else {
        prod *= f2;
        rank *= r2 + 1;
        doit = (rank < brows2);
      }
      if (doit) {
        int ind = mmhash2(r1, r2, nfeats);                     // Hash the indices
        if (transpose > 0) {
          float sum = A[threadIdx.x + nrows * i] * prod;    // Do the product
          atomicAdd(&C[threadIdx.x + nrows * ind], sum);
        } else {
          float sum = A[threadIdx.x + nrows * ind] * prod;  // Do the product
          atomicAdd(&C[threadIdx.x + nrows * i], sum);
        }
      }
    }
  }
}

__forceinline__ __device__ int hash2(int a, int b, int modulus) {
  return  (((a * 453423453) + b) * 34143242142) % modulus;
}

#if __CUDA_ARCH__ >= 300

// This version is designed for few (or one) row in A. It allocates one warp per column

__global__ void __hashmult2(int nrows, int nfeats, int ncols, int brows1, int brows2, float *A, float *Bdata, int *Bir, int *Bjc, float *C, int transpose) {
  bool doit = false;
  int istart = ((long long)blockIdx.x) * ncols / gridDim.x;
  int iend = ((long long)(blockIdx.x+1)) * ncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i];                                     // Range of nz rows in this column
    int jend = Bjc[i+1];
    int nr = jend - jstart;                                  // Number of nz rows
    for (int j1 = 0; j1 < nr; j1 += blockDim.x) {               // work on a block of data
      float f1 = 0;
      int r1 = -1;
      if (j1 + threadIdx.x < nr) {
        f1 = Bdata[jstart + j1 + threadIdx.x];                // Get the two features
        r1 = Bir[jstart + j1 + threadIdx.x];                  // And their row indices
      }
      for (int j2 = j1; j2 < nr; j2 += blockDim.x) {             // work on a block of data
        float f2 = 0;
        int r2 = -1;
        if (j2 + threadIdx.x < nr) {
          f2 = Bdata[jstart + j2 + threadIdx.x];
          r2 = Bir[jstart + j2 + threadIdx.x];
        }
        for (int k = 0; k < 32; k++) {
          float f2shift = __shfl(f2, k);
          int r2shift = __shfl(r2, k);
          if (j2 + k < nr && r1 >= 0) {
            long long rank = r1 + 1;
            float prod = f1;
            doit = false;
            if (j1 + threadIdx.x == j2 + k) {
              doit = (rank < brows1);
            } else if (j1 + threadIdx.x < j2 + k) {
              prod *= f2shift;
              rank *= r2shift + 1;
              doit = (rank < brows2);
            }
            if (doit) {
              int ind = mmhash2(r1, r2shift, nfeats);           // Hash the indices
              if (transpose > 0) {
                for (int m = 0; m < nrows; m++) {
                  float sum = A[m + nrows * i] * prod;    // Do the product
                  atomicAdd(&C[m + nrows * ind], sum);
                  //		  atomicAdd(&C[0], sum);
                }
              } else {
                for (int m = 0; m < nrows; m++) {
                  float sum = A[m + nrows * ind] * prod;  // Do the product
                  atomicAdd(&C[m + nrows * i], sum);
                  //		  atomicAdd(&C[0], sum);
                }
              }
            }
          }
        }
      }
    }
  }
}

#else

__global__ void __hashmult2(int nrows, int nfeats, int ncols, int brows1, int brows2, float *A, float *Bdata, int *Bir, int *Bjc, float *C, int transpose) {}

#endif

int hashmult(int nrows, int nfeats, int ncols, int brows1, int brows2, float *A, float *Bdata, int *Bir, int *Bjc, float *C, int transpose) {
  if (nrows >= 0) {
    int nt = max(1, 256/nrows);
    dim3 threadDim(nrows, nt, 1);
    int nblocks = min(MAXXGRID, ncols);
    __hashmult<<<nblocks,threadDim>>>(nrows, nfeats, ncols, brows1, brows2, A, Bdata, Bir, Bjc, C, transpose);
  } else {
    dim3 threadDim(32, 1, 1);
    int nblocks = min(MAXXGRID, ncols);
    __hashmult2<<<nblocks,threadDim>>>(nrows, nfeats, ncols, brows1, brows2, A, Bdata, Bir, Bjc, C, transpose);
  }
  hipStreamSynchronize(SYNC_STREAM);
  hipError_t err = hipGetLastError();
  return err;
}

//__forceinline__ __device__ long long __pairembed(long long r1, int r2) {
//  return ((r1+r2)*(r1+r2+1) >> 1) + r2;
//}

// The pair embedding function assumes r1x > r2x >= 0; 

__forceinline__ __device__ long long __pairembed(long long r1x, int r2x) {
  long long r1 = r1x+1;
  int r2 = r2x+1;
  float loc1 = (float) r1;
  float loc2 = (float) r2;
  int nbits1 = ((*(int *)(&loc1)) >> 23) - 126;
  int nbits2 = ((*(int *)(&loc2)) >> 23) - 126;
  int len = nbits1 + nbits2 - 2;
  float loc3 = (float) len; 
  int lenbits = 0;
  if (len > 1) lenbits = ((*(int *)(&loc3)) >> 23) - 127;
  r2 = r2 & ((1 << (nbits2-1)) - 1);
  long long x = (((r1 << (nbits2-1)) | r2) << lenbits) | (nbits2-1);
  return max((long long)0,x-2);
}

__global__ void __dopairembed(int *r1, int *r2, long long *res, int n) {
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < n; i += blockDim.x * gridDim.x * gridDim.y) {
    res[i] = __pairembed(r1[i], r2[i]);
  }
}

int pairembed(int *r1, int *r2, long long *res, int n) {
  int nthreads;
  dim3 griddims;
  setsizes(n, &griddims, &nthreads);
  __dopairembed<<<griddims,nthreads>>>(r1, r2, res, n);
  hipStreamSynchronize(SYNC_STREAM);
  hipError_t err = hipGetLastError();
  return err;
}


// Pair mult multplies base features and pairs of features.
//
// NOTE: The single-matrix version uses a fast lookup recurrence which is only valid up to 3000 base features per column (approx 4.5 million pairs)

// Given dense A and sparse B, for each column of B, enumerate all pairs of features, hash to a single feature index, and multiply by A into C
// todo: fix offsets

__global__ void __pairmult(int nrows, int bncols, int brows1, int brows2, float *A, int lda, float *A2, int lda2, 
                           float *Bdata, int *Bir, int *Bjc, int broff, int bcoff, float *C, int ldc, int transpose) {
  bool doit = false;
  int istart = ((long long)blockIdx.x) * bncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * bncols / gridDim.x;
  float *AX;
  int ldax;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i + bcoff];                             // Range of nz rows in this column
    int jend = Bjc[i+1 + bcoff];
    int nr = jend - jstart;                                  // Number of nz rows
    int todo = nr * (nr + 1) / 2;                            // Number of pairs to process (including k,k pairs)
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = solve1(j);                                    // Compute the first and second indices
      int j2 = j - j1*(j1+1)/2; 
      //      int j1, j2;
      //      solvex(todo, j, j1, j2);
      float f1 = Bdata[jstart + j1];                         // Get the two features
      float f2 = Bdata[jstart + j2];
      int r1 = Bir[jstart + j1] - broff;                             // And their row indices
      int r2 = Bir[jstart + j2] - broff;
      long long rank = r1;
      float prod = f1;
      doit = (r1 >= 0 && r2 >= 0);
      if (j1 == j2) {
        doit = doit && r1 < brows1;
        AX = A;
        ldax = lda;
      } else {
        rank = __pairembed(r1, r2);
        doit = doit && (rank >= 0 && rank < brows2);
        if (doit) {
          prod = f1*f2/(abs(f1)+abs(f2)+1.0e-7f);
          AX = A2;
          ldax = lda2;
        }
      }
      if (doit) {
        if (transpose > 0) {
          float sum = AX[threadIdx.x + ldax * i] * prod;    // Do the product
          atomicAdd(&C[threadIdx.x + ldc * rank], sum);
        } else {
          float sum = AX[threadIdx.x + ldax * rank] * prod;  // Do the product
          atomicAdd(&C[threadIdx.x + ldc * i], sum);
        }
      }
    }
  }
}

#if __CUDA_ARCH__ >= 300

// This version is designed for few (or one) rows in A. It allocates one warp per column
// todo: implement the offsets. 

__global__ void __pairmult2(int nrows, int bncols, int brows1, int brows2, float *A, int lda, float *A2, int lda2, 
                            float *Bdata, int *Bir, int *Bjc, int broff, int bcoff, float *C, int ldc, int transpose) {
  bool doit = false;
  int istart = ((long long)blockIdx.x) * bncols / gridDim.x;
  int iend = ((long long)(blockIdx.x+1)) * bncols / gridDim.x;
  float *AX;
  int ldax;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i + bcoff];                                     // Range of nz rows in this column
    int jend = Bjc[i+1 + bcoff];
    int nr = jend - jstart;                                  // Number of nz rows
    for (int j1 = 0; j1 < nr; j1 += blockDim.x) {               // work on a block of data
      float f1 = 0;
      int r1 = -1;
      if (j1 + threadIdx.x < nr) {
        f1 = Bdata[jstart + j1 + threadIdx.x];                // Get the two features
        r1 = Bir[jstart + j1 + threadIdx.x] - broff;                  // And their row indices
      }
      for (int j2 = j1; j2 < nr; j2 += blockDim.x) {             // work on a block of data
        float f2 = 0;
        int r2 = -1;
        if (j2 + threadIdx.x < nr) {
          f2 = Bdata[jstart + j2 + threadIdx.x];
          r2 = Bir[jstart + j2 + threadIdx.x] - broff;
        }
        for (int k = 0; k < 32; k++) {
          float f2shift = __shfl(f2, k);
          int r2shift = __shfl(r2, k);
          if (j2 + k < nr && r1 >= 0) {
            long long rank = r1;
            float prod = f1;
            doit = (r1 >= 0 && r1 < brows1 && r2 >= 0 && r2 < brows1);
            if (j1 + threadIdx.x == j2 + k) {
              AX = A;
              ldax = lda;
            } else if (j1 + threadIdx.x < j2 + k) {
              rank = __pairembed(r1, r2);
              doit = doit && (rank < brows2);
              if (doit) {
                prod *= f2shift;
                AX = A2;
                ldax = lda2;
              }
            }
            if (doit) {
              if (transpose > 0) {
                for (int m = 0; m < nrows; m++) {
                  float sum = AX[m + ldax * i] * prod;    // Do the product
                  atomicAdd(&C[m + ldc * rank], sum);
                  //		  atomicAdd(&C[0], sum);
                }
              } else {
                for (int m = 0; m < nrows; m++) {
                  float sum = AX[m + ldax * rank] * prod;  // Do the product
                  atomicAdd(&C[m + ldc * i], sum);
                  //		  atomicAdd(&C[0], sum);
                }
              }
            }
          }
        }
      }
    }
  }
}

#else

__global__ void __pairmult2(int nrows, int bncols, int brows1, int brows2, float *A, int lda, float *A2, int lda2, 
                            float *Bdata, int *Bir, int *Bjc, int broff, int bcoff, float *C, int ldc, int transpose) {}

#endif

int pairMultTile(int nrows, int bncols, int brows1, int brows2, float *A, int lda, float *A2, int lda2, 
                 float *Bdata, int *Bir, int *Bjc, int broff, int bcoff, float *C, int ldc, int transpose) {
  if (nrows >= 0) {
    int nt = max(1, 256/nrows);
    dim3 threadDim(nrows, nt, 1);
    int nblocks = min(MAXXGRID, bncols);
    __pairmult<<<nblocks,threadDim>>>(nrows, bncols, brows1, brows2, A, lda, A2, lda2, Bdata, Bir, Bjc, broff, bcoff, C, ldc, transpose);
  } else {
    dim3 threadDim(32, 1, 1);
    int nblocks = min(MAXXGRID, bncols);
    __pairmult2<<<nblocks,threadDim>>>(nrows, bncols, brows1, brows2, A, lda, A2, lda2, Bdata, Bir, Bjc, broff, bcoff, C, ldc, transpose);
  }
  hipStreamSynchronize(SYNC_STREAM);
  hipError_t err = hipGetLastError();
  return err;
}

__forceinline__ __device__ void __gupdate(float grad, int i, int ihere, int jhere, float *MM, float *Sumsq, float *Mask, int maskrows, float *lrate, int lrlen, 
                                              float *vexp, int vexplen, float *texp, int texplen, float istep, int addgrad, float epsilon) {
  float lr, ve, te, pve, ste, ngrad, ssq, ssqnew;
  ssq = Sumsq[ihere];
  ssqnew = hypotf(grad,ssq);
  atomicAdd(&Sumsq[ihere], ssqnew - ssq);
  ssq = ssqnew * sqrtf(istep);

  if (addgrad) {
    lr =  (lrlen > 1) ? lrate[i] : lrate[0];
    ve =  (vexplen > 1) ? vexp[i] : vexp[0];
    te =  (texplen > 1) ? texp[i] : texp[0];
    pve = (ve == 0.5f) ? ssq : ((ve == 0) ? 1.0f : pow(ssq, 2*ve));
    ste = pow(istep, te);
    ngrad = grad * lr * ste / pve;
    atomicAdd(&MM[ihere], ngrad);
  }
  if (Mask != NULL) {
    if (maskrows > 1) {
      if (Mask[ihere] == 0) MM[ihere] = 0;
    } else {
      if (Mask[jhere] == 0) MM[ihere] = 0;
    }
  }
}

/*
__forceinline__ __device__ void __gupdate(float grad, int i, int ithere, int jthere, float *MM, float *Sumsq, float *Mask, int maskrows, float *lrate, int lrlen, 
                                          float *vexp, int vexplen, float *texp, int texplen, float istep, int addgrad, float epsilon) {
  float lr, ve, te, pve, ste, ngrad;
  Sumsq[ithere] += grad * grad + epsilon;
  if (addgrad) {
    lr =  (lrlen > 1) ? lrate[i] : lrate[0];
    ve =  (vexplen > 1) ? vexp[i] : vexp[0];
    te =  (texplen > 1) ? texp[i] : texp[0];
    pve = (ve == 0) ? 1.0f : pow(Sumsq[ithere] * istep, ve);
    ste = pow(istep, te);
    ngrad = grad * lr * ste / pve;
    atomicAdd(&MM[ithere], ngrad);
  }
  if (Mask != NULL) {
    if (maskrows > 1) {
      if (Mask[ithere] == 0) MM[ithere] = 0;
    } else {
      if (Mask[jthere] == 0) MM[ithere] = 0;
    }
  }
  }*/

__global__ void __hashmultADAGrad(int nrows, int nfeats, int ncols, int brows1, int brows2, float *A, float *Bdata, int *Bir, int *Bjc, int transpose,
                                  float *MM, float *Sumsq, float *Mask, int maskrows, float *lrate, int lrlen, 
                                  float *vexp, int vexplen, float *texp, int texplen, float istep, int addgrad, float epsilon) {
  bool doit = false;
  int ihere, ithere, jthere;
  float grad;
  int istart = ((long long)blockIdx.x) * ncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * ncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i];                                     // Range of nz rows in this column
    int jend = Bjc[i+1];
    int nr = jend - jstart;                                  // Number of nz rows
    int todo = nr * (nr + 1) / 2;                            // Number of pairs to process (including k,k pairs)
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = solve1(j);                                    // Compute the first and second indices
      int j2 = j - j1*(j1+1)/2; 
      //      int j1, j2;
      //      solvex(todo, j, j1, j2);
      float f1 = Bdata[jstart + j1];                         // Get the two features
      float f2 = Bdata[jstart + j2];
      int r1 = Bir[jstart + j1];                             // And their row indices
      int r2 = Bir[jstart + j2];
      long long rank = r1 + 1;
      float prod = f1;
      if (j1 == j2) {
        doit = (rank < brows1);
      } else {
        prod *= f2;
        rank *= r2 + 1;
        doit = (rank < brows2);
      }
      if (doit) {
        int ind = mmhash2(r1, r2, nfeats);                     // Hash the indices
        if (transpose > 0) {
          ihere = threadIdx.x + nrows * i;
          ithere = threadIdx.x + nrows * ind;
          jthere = ind;
        } else {
          ithere = threadIdx.x + nrows * i;
          jthere = i;
          ihere = threadIdx.x + nrows * ind;
        }
        grad = A[ihere] * prod;    // raw gradient
        __gupdate(grad, threadIdx.x, ithere, jthere, MM, Sumsq, Mask, maskrows, lrate, lrlen, vexp, vexplen, texp, texplen, istep, addgrad, epsilon);
      }
    }
  }
}

int hashmultADAGrad(int nrows, int nfeats, int ncols, int brows1, int brows2, float *A, float *Bdata, int *Bir, int *Bjc, int transpose, 
                    float *MM, float *Sumsq, float *Mask, int maskrows, float *lrate, int lrlen, 
                    float *vexp, int vexplen, float *texp, int texplen, float istep, int addgrad, float epsilon) {
  int nt = max(1, 256/nrows);
  dim3 threadDim(nrows, nt, 1);
  int nblocks = min(MAXXGRID, ncols);
  __hashmultADAGrad<<<nblocks,threadDim>>>(nrows, nfeats, ncols, brows1, brows2, A, Bdata, Bir, Bjc, transpose, 
                                           MM, Sumsq, Mask, maskrows, lrate, lrlen, vexp, vexplen, texp, texplen, istep, addgrad, epsilon);
  hipStreamSynchronize(SYNC_STREAM);
  hipError_t err = hipGetLastError();
  return err;
}
//
// nrows = rows of MM (and other model mats)
// ncols = columns of B = columns of A 
//
__global__ void __pairMultADAGradTile(int nrows, int bncols, int brows1, int brows2, float *A, int lda, int aroff, int acoff, 
                                      float *Bdata, int *Bir, int *Bjc, int broff, int bcoff, int transpose,
                                      float *MM, int ldmm, float *Sumsq, float *Mask, int maskrows, float *lrate, int lrlen, 
                                      float *vexp, int vexplen, float *texp, int texplen, float istep, int addgrad, float epsilon) {
  bool doit = false;
  int ihere, ithere, jhere, jthere;
  float grad;
  int istart = ((long long)blockIdx.x) * bncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * bncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart = Bjc[i+bcoff];                               // Range of nz rows in this column
    int jend = Bjc[i+1+bcoff];
    int nr = jend - jstart;                                  // Number of nz rows
    int todo = nr * (nr + 1) / 2;                            // Number of pairs to process (including k,k pairs)
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = solve1(j);                                    // Compute the first and second indices
      int j2 = j - j1*(j1+1)/2; 
      //      int j1, j2;
      //      solvex(todo, j, j1, j2);
      float f1 = Bdata[jstart + j1];                         // Get the two features
      float f2 = Bdata[jstart + j2];
      int r1 = Bir[jstart + j1]-broff;                       // And their row indices
      int r2 = Bir[jstart + j2]-broff;
      long long rank = r1;
      float prod = f1;
      doit = (r1 >= 0 && r2 >= 0);
      if (doit) {
        if (j1 == j2) {
          doit = doit && r1 < brows1;
          ithere = 0;
          jthere = 0;
        } else {
          rank = __pairembed(r1, r2);
          doit = doit && (rank < brows2);
          if (doit) {
            prod = f1*f2/(abs(f1)+abs(f2)+1.0e-7f);
            ithere = ldmm;
            jthere = 1;
          }
        }
      }
      if (doit) {
        if (transpose > 0) {
          ihere = threadIdx.x + aroff + lda * (i + acoff);
          jhere = threadIdx.x + aroff;
          ithere += threadIdx.x + 2 * ldmm * rank;
          jthere += 2 * rank;
        } else {
          ihere = threadIdx.x + aroff + lda * (rank + acoff);
          jhere = threadIdx.x + aroff;
          ithere += threadIdx.x + 2 * ldmm * i;
          jthere += 2 * i;
        }
        grad = A[ihere] * prod;    // raw gradient
        __gupdate(grad, jhere, ithere, jthere, MM, Sumsq, Mask, maskrows, lrate, lrlen, vexp, vexplen, texp, texplen, istep, addgrad, epsilon);
      }
    }
  }
}

int pairMultADAGradTile(int nrows, int bncols, int brows1, int brows2, float *A, int lda, int aroff, int acoff,
                        float *Bdata, int *Bir, int *Bjc, int broff, int bcoff, int transpose, 
                        float *MM, int ldmm, float *Sumsq, float *Mask, int maskrows, float *lrate, int lrlen, 
                        float *vexp, int vexplen, float *texp, int texplen, float istep, int addgrad, float epsilon) {
  int nt = max(1, 256/nrows);
  dim3 threadDim(nrows, nt, 1);
  int nblocks = min(MAXXGRID, bncols);
  __pairMultADAGradTile<<<nblocks,threadDim>>>(nrows, bncols, brows1, brows2, A, lda, aroff, acoff, Bdata, Bir, Bjc, broff, bcoff, transpose, 
                                               MM, ldmm, Sumsq, Mask, maskrows, lrate, lrlen, vexp, vexplen, texp, texplen, istep, addgrad, epsilon);
  hipStreamSynchronize(SYNC_STREAM);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __hashcross(int nrows, int nfeats, int ncols,
			     float *A,
			     float *Bdata, int *Bir, int *Bjc,
			     float *Cdata, int *Cir, int *Cjc,
			     float *D, int transpose) {
  int r1, r2, ind;
  int istart = ((long long)blockIdx.x) * ncols/ gridDim.x;
  int iend = ((long long)(blockIdx.x + 1)) * ncols / gridDim.x;
  for (int i = istart; i < iend ; i++) {                     // i is the column index
    int jstart1 = Bjc[i];                                    // Range of nz rows in this column of B
    int jend1 = Bjc[i+1];
    int jstart2 = Cjc[i];                                    // Range of nz rows in this column of C
    int jend2 = Cjc[i+1];
    int nr1 = jend1 - jstart1;                               // Number of nz rows
    int nr2 = jend2 - jstart2;                               // Number of nz rows
    int todo = (nr1+1) * (nr2+1) - 1;                        // Number of pairs + singletons to process 
    for (int j = threadIdx.y; j < todo; j += blockDim.y) {   // j indexes a worker for this column
      int j1 = j / nr2;
      int j2 = j - j1 * nr2; 
      float prod = 1.0f;
      int hash = seed;
      if (j1 < nr1) {
        prod *= Bdata[jstart1 + j1];                         // Get the two features
        r1 = Bir[jstart1 + j1];                              // And their row indices
        hash = h1(r1, hash);
      }
      if (j2 < nr2) {
        prod *= Cdata[jstart2 + j2];
        r2 = Cir[jstart2 + j2];
        hash = h1(r2, hash);                                 // Hash the indices
      } 
      ind = mmhashend(hash, nfeats);
      if (transpose > 0) {
        float sum = A[threadIdx.x + nrows * i] * prod;       // Do the product
        atomicAdd(&D[threadIdx.x + nrows * ind], sum);
      } else {
        float sum = A[threadIdx.x + nrows * ind] * prod;     
        atomicAdd(&D[threadIdx.x + nrows * i], sum);
      }
    }
  }
}

int hashcross(int nrows, int nfeats, int ncols, float *A, float *Bdata, int *Bir, int *Bjc, float *Cdata, int *Cir, int *Cjc, float *D, int transpose) {
  int nt = max(1, 256/nrows);
  dim3 threadDim(nrows, nt, 1);
  int nblocks = min(MAXXGRID, ncols);
  __hashcross<<<nblocks,threadDim>>>(nrows, nfeats, ncols, A, Bdata, Bir, Bjc, Cdata, Cir, Cjc, D, transpose);
  hipStreamSynchronize(SYNC_STREAM);
  hipError_t err = hipGetLastError();
  return err;
}
