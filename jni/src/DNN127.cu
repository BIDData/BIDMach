#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <MatKernel.hpp>

#define BYDIMF 5
#define CDIM 10

#define BYDIMB 5


#if __CUDA_ARCH__ >= 300

/*
 * Positive kernel for word2vec. This handles the positively-label word pairs with
 * one context word and the current word. 
 */


template<int SKIP, int YDIM, int NREPS>
  __global__ void __word2vecPos(int nrows, int ncols, int *W, int *LB, int *UB, float *A, float *B, float lrate, float vexp) {
  const int nwindow = 2*SKIP+1; 
  int iwords[nwindow];
  float aa[NREPS];
  float daa[NREPS];
  float bb[NREPS][nwindow];
  float dbb[NREPS][nwindow];
  __shared__ float CC[YDIM * nwindow];

  int i, j, k, tid, indx, icol, dxy, lb, ub;
  float prod, v, ascale, bscale;
  tid = threadIdx.x + blockDim.x * threadIdx.y;
  dxy = blockDim.x * blockDim.y;
  bool good;

  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);
  float inr = 1.0f / nrows;

#pragma unroll
  for (i = 0; i < nwindow; i++) {                           // Prefill the word and aa window buffers
    if (istart + i - SKIP - 1 >= 0) {
      iwords[i] = nrows * W[istart + i - SKIP - 1];         // Get a new word address
    } else {
      iwords[i] = -1;
    }
    good = (iwords[i] >= 0);
#pragma unroll
    for (j = 0; j < NREPS; j++) {                           // Get the B vector for this word
      indx = tid + j * dxy;
      if (good && indx < nrows) {
        bb[j][i] = B[indx + iwords[i]];
      } else {
        bb[j][i] = 0;
      }
      dbb[j][i] = 0;
    }
  }

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns
#pragma unroll
    for (i = 0; i < nwindow-1; i++) {                       // slide iwords down
      iwords[i] = iwords[i+1];
#pragma unroll
      for (j = 0; j < NREPS; j++) {
        bb[j][i] = bb[j][i+1];                              // slide data down
        dbb[j][i] = dbb[j][i+1];                            // slide deriv down
      }
    }

    good = (icol + SKIP < ncols);
    if (good) {
      iwords[nwindow - 1] = nrows * W[icol + SKIP];         // Get a new word address
    } else {
      iwords[nwindow - 1] = -1;
    }
    good = good && iwords[nwindow-1] >= 0;

#pragma unroll
    for (j = 0; j < NREPS; j++) {                           // Get a new B column
      indx = tid + j * dxy;
      if (good && indx < nrows) {
        bb[j][nwindow - 1] = B[indx + iwords[nwindow - 1]];
      } else {
        bb[j][nwindow - 1] = 0;
      }
      dbb[j][nwindow-1] = 0;
      if (iwords[SKIP] >= 0 && indx < nrows) {               // Get a new A column
        aa[j] = A[indx + iwords[SKIP]];
      } else {
        aa[j] = 0;
      }
    }
    lb = LB[icol];
    ub = UB[icol];

    __syncthreads();
    if (iwords[SKIP] >= 0) {
#pragma unroll                 
      for (i = 0; i < nwindow; i++) {                         // Iterate across the window for B cols
        prod = 0;
        if (i >= SKIP + lb && i <= SKIP + ub && i != SKIP) {
#pragma unroll                 
          for (j = 0; j < NREPS; j++) {                       // Iterate over blocks of elements
            prod += bb[j][i] * aa[j];                         // Compute the product between current A, B cols
          }
#pragma unroll                 
          for (k = 1; k < 32; k = k + k) {
            v = __shfl_down(prod, k);                         // Reduce within warp
            prod += v;
          }  
          if (threadIdx.x == 0) {
            CC[i - SKIP - lb + threadIdx.y * nwindow] = prod;  // Save to SHMEM
          }
        }
      }

      __syncthreads();
      for (i = 1; i < blockDim.y; i++) {                      // Reduce across warps
        for (k = tid; k <= ub - lb; k += dxy) { 
          CC[k] += CC[k + i * nwindow];
        }
        __syncthreads();
      }

      __syncthreads();                                        //  Apply the sigmoid map
      for (i = tid; i <= ub - lb; i += dxy) { 
        v = CC[i];
        if (v > 16.0f) {
          v = 1.0f;
        } else if (v < -16.0f) {
          v = 0.0f;
        } else {
          v = exp(v);
          v = v / (1.0f + v);
        }
        CC[i] = 1.0f - v;                                     // All pairs have label 1
      }
      
      __syncthreads();  
#pragma unroll                 
      for (j = 0; j < NREPS; j++) {
        daa[j] = 0;
      }
      ascale = pow(max(0, iwords[SKIP])*inr + 1.0f, vexp);
#pragma unroll                 
      for (i = 0; i < nwindow; i++) {                         // Iterate across the window for A cols
        if (i >= SKIP + lb && i <= SKIP + ub && i != SKIP && iwords[i] >= 0) {
          bscale = pow(max(0, iwords[i])*inr + 1.0f, vexp);
          v = lrate * CC[i - SKIP - lb];
#pragma unroll                 
          for (j = 0; j < NREPS; j++) {
            daa[j] += ascale * v * bb[j][i];                           // Update A's derivative
            dbb[j][i] += bscale * v * aa[j];                           // Update B's derivative
          }
        }
      }
      __syncthreads();  
#pragma unroll                 
      for (j = 0; j < NREPS; j++) { 
        if (tid + j * dxy < nrows) {                        // Save the A column
          atomicAdd(&A[tid + j * dxy + iwords[SKIP]], daa[j]);
        }
      } 
      if (iwords[0] >= 0) {
#pragma unroll                 
        for (j = 0; j < NREPS; j++) { 
          if (tid + j * dxy < nrows) {                        // Save the B column
            atomicAdd(&B[tid + j * dxy + iwords[0]], dbb[j][0]);
          }
        }
      }
      __syncthreads();  
    }
  }

#pragma unroll      
  for (i = 1; i < nwindow; i++) {                           // Clear out the derivative queue
    if (iwords[i] >= 0) {
#pragma unroll                 
      for (j = 0; j < NREPS; j++) {                         // Save the B column
        if (tid + j * dxy < nrows) {
          atomicAdd(&B[tid + j * dxy + iwords[i]], dbb[j][i]);
        }
      } 
    }
  }
}

/*
 * Convolutional kernel for word2vec. This handles the positively-label word pairs with
 * one context word and the current word. 
 */

template<int SKIP, int YDIM, int NREPS>
  __global__ void __word2vecEvalPos(int nrows, int ncols, int *W, int *LB, int *UB, float *A, float *B, float *Retval) {
  const int nwindow = 2*SKIP+1; 
  int iwords[nwindow];
  float aa[NREPS];
  float bb[NREPS][nwindow];
  __shared__ float CC[YDIM * nwindow];

  int i, j, k, tid, indx, icol, dxy, lb, ub;
  float prod, v;
  tid = threadIdx.x + blockDim.x * threadIdx.y;
  dxy = blockDim.x * blockDim.y;
  bool good;
  double sum = 0;

  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

#pragma unroll
  for (i = 0; i < nwindow; i++) {                           // Prefill the word and aa window buffers
    if (istart + i - SKIP - 1 >= 0) {
      iwords[i] = nrows * W[istart + i - SKIP - 1];          // Get a new word
    } else {
      iwords[i] = -1;
    }
    good = (iwords[i] >= 0);
#pragma unroll
    for (j = 0; j < NREPS; j++) {                           // Get the B vector for this word
      indx = tid + j * dxy;
      if (good && indx < nrows) {
        bb[j][i] = B[indx + iwords[i]];
      } else {
        bb[j][i] = 0;
      }
    }
  }

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns
#pragma unroll
    for (i = 0; i < nwindow-1; i++) {                       // slide iwords down
      iwords[i] = iwords[i+1];
#pragma unroll
      for (j = 0; j < NREPS; j++) {
        bb[j][i] = bb[j][i+1];                              // slide data down
      }
    }

    good = (icol + SKIP < ncols);
    if (good) {
      iwords[nwindow - 1] = nrows * W[icol + SKIP];          // Get a new word
    } else {
      iwords[nwindow - 1] = -1;
    }
    good = good && iwords[nwindow-1] >= 0;

#pragma unroll
    for (j = 0; j < NREPS; j++) {                           // Get a new B column
      indx = tid + j * dxy;
      if (good && indx < nrows) {
        bb[j][nwindow - 1] = B[indx + iwords[nwindow - 1]];
      } else {
        bb[j][nwindow - 1] = 0;
      }
      if (iwords[SKIP] >= 0 && indx < nrows) {               // Get a new A column
        aa[j] = A[indx + iwords[SKIP]];
      } else {
        aa[j] = 0;
      }
    }
    lb = LB[icol];
    ub = UB[icol];

    __syncthreads();
#pragma unroll                 
    for (i = 0; i < nwindow; i++) {                           // Iterate across the window for B cols
      if (i >= SKIP + lb && i <= SKIP + ub) {
        if (i == SKIP || iwords[SKIP] < 0 || iwords[i] < 0) { // Give this word a large score (gives zero contribution to loss)
          prod = 20.0f;
        } else {
          prod = 0;
#pragma unroll                 
          for (j = 0; j < NREPS; j++) {                       // Iterate over blocks of elements
            prod += bb[j][i] * aa[j];                         // Compute the product between current A, B cols
          }
#pragma unroll                 
          for (k = 1; k < 32; k = k + k) {
            v = __shfl_down(prod, k);                         // Reduce within warp
            prod += v;
          }  
        }
        if (threadIdx.x == 0) {
          CC[i - SKIP - lb + threadIdx.y * nwindow] = prod;  // Save to SHMEM
        }
      }
    }

    __syncthreads();
    for (i = 1; i < blockDim.y; i++) {                      // Reduce across warps
      for (k = tid; k <= ub - lb; k += dxy) { 
        CC[k] += CC[k + i * nwindow];
      }
      __syncthreads();
    }

    __syncthreads();                                        //  Apply the sigmoid map
    for (i = tid; i <= ub - lb; i += dxy) { 
      v = CC[i];
      if (v > 16.0f) {
        v = 1.0f;
      } else if (v < -16.0f) {
        v = 0.0f;
      } else {
        v = exp(v);
        v = v / (1.0f + v);
      }
      CC[i] = log(max(v, 1.0e-20f));                      // Compute the loss
    }
    __syncthreads();
    for (i = 1; i <= ub - lb; i = i + i) {
      if ((tid & (i-1)) == 0 && tid + i <= ub - lb) {
        CC[tid] += CC[tid + i];
      }
      __syncthreads();
    }
    sum += CC[0];
    __syncthreads();
  }
  if (tid == 0) {
    atomicAdd(&Retval[0], (float)sum);
  }
}

template<int NSKIP, int BYDIM>
  __global__ void __word2vecPosy(int nrows, int ncols, int *W,  int *LB, int *UB, float *A, float *B, float lrate, float vexp) {
  __shared__ float CC[NSKIP*2*BYDIM];
  float aa;
  int ib[NSKIP*2];
  float prods[NSKIP*2];
  float bscale[NSKIP*2];
  int ia, iword, lb, ub;
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);
  int i, j, k, icol, jcol;
  float bb, db, dv, v, ascale, tmp;
  float inr = 1.0f / nrows;

  for (icol = istart; icol < iend; icol++) {                          // Iterate over columns
    ia = nrows * W[icol];   
    if (ia >= 0) {                                                    // Load lb and ub values
      lb = LB[icol];
      ub = UB[icol];
      jcol = threadIdx.x - NSKIP;
      iword = -1;
      if (jcol >= lb && jcol <= ub) {                                 // Load words in the window
        iword = W[icol + jcol];
      }
#pragma unroll
      for (i = 0; i < NSKIP; i++) {                                   // Share window word ids across threads, clear prods
        ib[i] = nrows * __shfl(iword, i);
        ib[i+NSKIP] = nrows * __shfl(iword, i+NSKIP+1);
        prods[i] = 0;
        prods[i+NSKIP] = 0;
      }

      for (i = tid; i < nrows; i += dxy) {                            // Compute products between center and context words
        aa = A[i + ia];
#pragma unroll
        for (j = 0; j < NSKIP*2; j++) {
          if (ib[j] >= 0) {
            bb = B[i + ib[j]];
            prods[j] += aa * bb;
          }
        }
      }         
                                              
#pragma unroll
      for (j = 0; j < NSKIP*2; j++) {                                 // Reduce prods within each warp
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          tmp = __shfl_down(prods[j], k);
          prods[j] += tmp;
        }
      }
      __syncthreads();

      if (threadIdx.x == 0) {                                         // Save the products to SHMEM (one copy per warp)
#pragma unroll
        for (j = 0; j < 2*NSKIP; j++) {
          CC[j + NSKIP * 2 * threadIdx.y] = prods[j];
        }
      }
      __syncthreads();

      for (i = 1; i < blockDim.y; i++) {                              // Reduce the products across warps
        __syncthreads();
        for (j = tid; j < NSKIP * 2; j += dxy) {
          CC[j] += CC[j + i * NSKIP * 2];
        } 
      } 
      __syncthreads();

      for (i = tid; i < NSKIP * 2; i+= dxy) {                         // Compute logistic function on all products
        v = CC[i];
        if (v > 16.0f) {
          v = 1.0f;
        } else if (v < -16.0f) {
          v = 0.0f;
        } else {
          v = exp(v);
          v = v / (1.0f + v);
        }
        CC[i] = lrate * (1 - v);                                      // All these pairs have label 1
      }
      __syncthreads();                                                // Now do scaled gradients

      ascale = pow(max(0, ia)*inr + 1.0f, vexp);                      // Simulated ADAGRAD on A
      for (j = 0; j < NSKIP * 2; j++) {                               // Load B data
        if (ib[j] >= 0) {
          bscale[j] = pow(max(0, ib[j])*inr + 1.0f, vexp);            // Simulated ADAGRAD on B
        } else {
          bscale[j] = 0;
        }
        prods[j] = CC[j];
      }
      __syncthreads();

      dv = 0;
      for (i = tid; i < nrows; i += dxy) {                            // Update vecs with derivatives
        aa = A[i + ia];
#pragma unroll
        for (j = 0; j < NSKIP * 2; j++) {                             // Load B data
          if (ib[j] >= 0) {
            bb = B[i + ib[j]];
            dv += ascale * prods[j] * bb;
            db = bscale[j] * prods[j] * aa;
            atomicAdd(&B[i + ib[j]], db);                             // Update B
          }
        }
        atomicAdd(&A[i + ia], dv);                                    // Update A
      } 
      __syncthreads();
    }
  }
}

template<int NSKIP, int BYDIM>
  __global__ void __word2vecEvalPosy(int nrows, int ncols, int *W,  int *LB, int *UB, float *A, float *B, float *retval) {
  __shared__ float CC[NSKIP*2*BYDIM];
  float aa;
  float prods[NSKIP*2];
  int ia, iword, lb, ub;
  int ib[NSKIP*2];
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);
  int i, j, k, icol, jcol;
  float bb, v, tmp, sum;

  sum = 0;
  for (icol = istart; icol < iend; icol++) {                          // Iterate over columns
    ia = nrows * W[icol];   
    if (ia >= 0) {                                                    // Load lb and ub values
      lb = LB[icol];
      ub = UB[icol];
      jcol = threadIdx.x - NSKIP;
      iword = -1;
      if (jcol >= lb && jcol <= ub) {                                 // Load words in the window
        iword = W[icol + jcol];
      }
#pragma unroll
      for (i = 0; i < NSKIP; i++) {                                   // Share window word ids across threads, clear prods
        ib[i] = nrows * __shfl(iword, i);
        ib[i+NSKIP] = nrows * __shfl(iword, i+NSKIP+1);
        prods[i] = 0;
        prods[i+NSKIP] = 0;
      }

      for (i = tid; i < nrows; i += dxy) {                            // Compute products between center and context words
        aa = A[i + ia];
#pragma unroll
        for (j = 0; j < NSKIP*2; j++) {
          if (ib[j] >= 0) {
            bb = B[i + ib[j]];
            prods[j] += aa * bb;
          }
        }
      }         
                                              
#pragma unroll
      for (j = 0; j < NSKIP*2; j++) {                                 // Reduce prods within each warp
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          tmp = __shfl_down(prods[j], k);
          prods[j] += tmp;
        }
      }
      __syncthreads();

      if (threadIdx.x == 0) {                                         // Save the products to SHMEM (one copy per warp)
#pragma unroll
        for (j = 0; j < 2*NSKIP; j++) {
          CC[j + NSKIP * 2 * threadIdx.y] = prods[j];
        }
      }
      __syncthreads();

      for (i = 1; i < blockDim.y; i++) {                              // Reduce the products across warps
        __syncthreads();
        for (j = tid; j < NSKIP * 2; j += dxy) {
          CC[j] += CC[j + i * NSKIP * 2];
        } 
      } 
      __syncthreads();

      for (i = tid; i < NSKIP * 2; i+= dxy) {                         // Compute logistic function on all products
        v = CC[i];
        if (v > 16.0f) {
          v = 1.0f;
        } else if (v < -16.0f) {
          v = 0.0f;
        } else {
          v = exp(v);
          v = v / (1.0f + v);
        }
        CC[i] = log(max(v, 1.0e-20f));                                // All these pairs have label 1
      }

      __syncthreads();                                                // Now sum likelihood over window 
      for (i = 1; i < 2 * NSKIP; i = i + i) {
        if ((tid & (i-1)) == 0 && tid + i < 2 * NSKIP) {
          CC[tid] += CC[tid + i];
        }
        __syncthreads();
      }
      sum += CC[0];
      __syncthreads();
    }
  }
  if (tid == 0) {
    atomicAdd(&retval[0], (float)sum);
  }
}


/*
 * Combined forward-backward word2vec kernel
 */

template<int NWA, int NWB, int BYDIM>
  __global__ void __word2vecNeg(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float lrate, float vexp) {
  const int NWAB = NWA*NWB;
  __shared__ float CC[NWA*NWB*BYDIM];
  float aa[NWA];
  float bb[NWB];
  float prods[NWA][NWB];
  int ia[NWA];
  int ib[NWB];
  float bscale[NWB];
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);
  int i, j, k, icol;
  float dv, v, ascale;
  float inr = 1.0f / nrows;

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns
#pragma unroll
    for (i = 0; i < NWA; i++) {
      ia[i] = nrows * WA[i + icol * NWA];                   // Fill the A word matrix
#pragma unroll
      for (j = 0; j < NWB; j++) {                           // clear the products matrix
        prods[i][j] = 0;
      }
    }
#pragma unroll
    for (i = 0; i < NWB; i++) {
      ib[i] = nrows * WB[i + icol * NWB];                   // Fill the B word matrix
    }

    for (i = tid; i < nrows; i += dxy) {                    // Now iterate over the rows of this block
#pragma unroll
      for (j = 0; j < NWB ; j++) {                          // Read B
        bb[j] = B[i + ib[j]];
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Compute the products of these elements
        v = A[i + ia[j]];
#pragma unroll
        for (k = 0; k < NWB; k++) {
          prods[j][k] += v * bb[k];
        }
      }
    }                                                       // Finished the entire block

#pragma unroll
    for (i = 0; i < NWA; i++) {                             // Reduce the products within each warp
#pragma unroll
      for (j = 0; j < NWB; j++) {
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          float tmp = __shfl_down(prods[i][j], k);
          prods[i][j] += tmp;
        }
      }
    }
    __syncthreads();
    if (threadIdx.x == 0) {                                 // Save the products to SHMEM (one copy per warp)
#pragma unroll
      for (i = 0; i < NWA; i++) {
#pragma unroll
        for (j = 0; j < NWB; j++) {
          CC[i + NWA * (j + NWB * threadIdx.y)] = prods[i][j];
        }
      }
    }
    __syncthreads();
    for (i = 1; i < blockDim.y; i++) {
      __syncthreads();
      for (j = tid; j < NWAB; j += dxy) {                   // Reduce the products across warps
        CC[j] += CC[j + i * NWAB];
      } 
    } 
    __syncthreads();

    for (i = tid; i < NWA*NWB; i+= dxy) {                   // Compute logistic function on all products
      v = CC[i];
      if (v > 16.0f) {
        v = 1.0f;
      } else if (v < -16.0f) {
        v = 0.0f;
      } else {
        v = exp(v);
        v = v / (1.0f + v);
      }
      CC[i] = - lrate * v;                                  // All these pairs have label 0
    }

    __syncthreads();
    for (i = tid; i < nrows; i += dxy) {
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Load A data
        aa[j] = A[i + ia[j]];
      }
#pragma unroll
      for (k = 0; k < NWB; k++) {                           // Load B data
        bb[k] = B[i + ib[k]];
        bscale[k] = pow(max(0, ib[k])*inr + 1.0f, vexp);
        prods[0][k] = 0;
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Now do the products
        ascale = pow(max(0, ia[j])*inr + 1.0f, vexp);
        dv = 0;
#pragma unroll
        for (k = 0; k < NWB; k++) {                       
          v = CC[j + k * NWA];
          dv += ascale * v * bb[k];
          prods[0][k] += bscale[k] * v * aa[j];
        }
        atomicAdd(&A[i + ia[j]], dv);                      // Update A
      }
#pragma unroll
      for (k = 0; k < NWB; k++) {                       
        atomicAdd(&B[i + ib[k]], prods[0][k]);             // Update B
      }
    } 
    __syncthreads();
  }
}


template<int NWA, int NWB, int BYDIM>
  __global__ void __word2vecNegFilt(int nrows, int ncols, int nwords, int *WA, int *WB, float *A, float *B, float lrate, float vexp) {
  const int NWAB = NWA*NWB;
  __shared__ float CC[NWA*NWB*BYDIM];
  float aa[NWA];
  float bb[NWB];
  float prods[NWA][NWB];
  int ia[NWA];
  int ib[NWB];
  float bscale[NWB];
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);
  int i, j, k, icol, tmpi;
  float dv, v, ascale;
  float inr = 1.0f / nrows;

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns
#pragma unroll
    for (i = 0; i < NWA; i++) {
      tmpi = WA[i + icol * NWA];                            // Fill the A word matrix
      if (tmpi < nwords) {
        tmpi = nrows * tmpi;
      } else {
        tmpi = -1;
      }
      ia[i] = tmpi;
#pragma unroll
      for (j = 0; j < NWB; j++) {                           // clear the products matrix
        prods[i][j] = 0;
      }
    }
#pragma unroll
    for (i = 0; i < NWB; i++) {
      tmpi = WB[i + icol * NWB];                            // Fill the B word matrix
      if (tmpi < nwords) {
        tmpi = nrows * tmpi;
      } else {
        tmpi = -1;
      }
      ib[i] = tmpi;
    }

    for (i = tid; i < nrows; i += dxy) {                    // Now iterate over the rows of this block
#pragma unroll
      for (j = 0; j < NWB ; j++) {                          // Read B
        if (ib[j] >= 0) {
          bb[j] = B[i + ib[j]];
        } else {
          bb[j] = 0;
        }
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Compute the products of these elements
        if (ia[j] >= 0) {
          v = A[i + ia[j]];
        } else {
          v = 0;
        }
#pragma unroll
        for (k = 0; k < NWB; k++) {
          prods[j][k] += v * bb[k];
        }
      }
    }                                                       // Finished the entire block

#pragma unroll
    for (i = 0; i < NWA; i++) {                             // Reduce the products within each warp
#pragma unroll
      for (j = 0; j < NWB; j++) {
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          float tmp = __shfl_down(prods[i][j], k);
          prods[i][j] += tmp;
        }
      }
    }
    __syncthreads();
    if (threadIdx.x == 0) {                                 // Save the products to SHMEM (one copy per warp)
#pragma unroll
      for (i = 0; i < NWA; i++) {
#pragma unroll
        for (j = 0; j < NWB; j++) {
          CC[i + NWA * (j + NWB * threadIdx.y)] = prods[i][j];
        }
      }
    }
    __syncthreads();
    for (i = 1; i < blockDim.y; i++) {
      __syncthreads();
      for (j = tid; j < NWAB; j += dxy) {                   // Reduce the products across warps
        CC[j] += CC[j + i * NWAB];
      } 
    } 
    __syncthreads();

    for (i = tid; i < NWA*NWB; i+= dxy) {                   // Compute logistic function on all products
      v = CC[i];
      if (v > 16.0f) {
        v = 1.0f;
      } else if (v < -16.0f) {
        v = 0.0f;
      } else {
        v = exp(v);
        v = v / (1.0f + v);
      }
      CC[i] = - lrate * v;                                  // All these pairs have label 0
    }

    __syncthreads();
    for (i = tid; i < nrows; i += dxy) {
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Load A data
        if (ia[j] >= 0) {
          aa[j] = A[i + ia[j]];
        } else {
          aa[j] = 0;
        }
      }
#pragma unroll
      for (k = 0; k < NWB; k++) {                           // Load B data
        if (ib[k] >= 0) {
          bb[k] = B[i + ib[k]];
        } else {
          bb[k] = 0;
        }
        bscale[k] = pow(max(0, ib[k])*inr + 1.0f, vexp);
        prods[0][k] = 0;
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Now do the products
        ascale = pow(max(0, ia[j])*inr + 1.0f, vexp);
        dv = 0;
#pragma unroll
        for (k = 0; k < NWB; k++) {                       
          v = CC[j + k * NWA];
          dv += ascale * v * bb[k];
          prods[0][k] += bscale[k] * v * aa[j];
        }
        if (ia[j] >= 0) {
          atomicAdd(&A[i + ia[j]], dv);                      // Update A
        }
      }
#pragma unroll
      for (k = 0; k < NWB; k++) {                       
        if (ib[k] >= 0) {
          atomicAdd(&B[i + ib[k]], prods[0][k]);             // Update B
        }
      }
    } 
    __syncthreads();
  }
}


/*
 * Combined forward-backward word2vec kernel
 */


template<int NWA, int NWB, int BYDIM>
  __global__ void __word2vecEvalNeg(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *Retval) {
  const int NWAB = NWA*NWB;
  __shared__ float CC[NWA*NWB*BYDIM];
  float bb[NWB];
  float prods[NWA][NWB];
  int ia[NWA];
  int ib[NWB];
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);
  int i, j, k, icol;
  float v;
  double sum = 0;

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns
#pragma unroll
    for (i = 0; i < NWA; i++) {
      ia[i] = nrows * WA[i + icol * NWA];                   // Fill the A word matrix
#pragma unroll
      for (j = 0; j < NWB; j++) {                           // clear the products matrix
        prods[i][j] = 0;
      }
    }
#pragma unroll
    for (i = 0; i < NWB; i++) {
      ib[i] = nrows * WB[i + icol * NWB];                   // Fill the B word matrix
    }

    for (i = tid; i < nrows; i += dxy) {                    // Now iterate over the rows of this block
#pragma unroll
      for (j = 0; j < NWB ; j++) {                          // Read B
        bb[j] = B[i + ib[j]];
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Compute the products of these elements
        v = A[i + ia[j]];
#pragma unroll
        for (k = 0; k < NWB; k++) {
          prods[j][k] += v * bb[k];
        }
      }
    }                                                       // Finished the entire block

#pragma unroll
    for (i = 0; i < NWA; i++) {                             // Reduce the products within each warp
#pragma unroll
      for (j = 0; j < NWB; j++) {
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          float tmp = __shfl_down(prods[i][j], k);
          prods[i][j] += tmp;
        }
      }
    }
    __syncthreads();
    if (threadIdx.x == 0) {                                 // Save the products to SHMEM (one copy per warp)
#pragma unroll
      for (i = 0; i < NWA; i++) {
#pragma unroll
        for (j = 0; j < NWB; j++) {
          CC[i + NWA * (j + NWB * threadIdx.y)] = prods[i][j];
        }
      }
    }
    __syncthreads();
    for (i = 1; i < blockDim.y; i++) {
      __syncthreads();
      for (j = tid; j < NWAB; j += dxy) {                   // Reduce the products across warps
        CC[j] += CC[j + i * NWAB];
      } 
    } 
    __syncthreads();

    for (i = tid; i < NWA*NWB; i+= dxy) {                   // Compute logistic function on all products
      v = CC[i];
      if (v > 16.0f) {
        v = 1.0f;
      } else if (v < -16.0f) {
        v = 0.0f;
      } else {
        v = exp(v);
        v = v / (1.0f + v);
      }
      CC[i] = log(max(1.0f - v, 1.0e-20f));                  // All these pairs have label 0
    }
    for (i = 1; i < NWA*NWB; i = i + i) {
      if ((tid & (i-1)) == 0 && tid + i < NWA*NWB) {
        CC[tid] += CC[tid + i];
      }
      __syncthreads();
    }
    sum += CC[0];
    __syncthreads();
  }
  if (tid == 0) {
    atomicAdd(&Retval[0], (float)sum);
  }
}

/*
 * Convolutional kernel for word2vec. This handles the positively-label word pairs with
 * one context word and the current word. 
 */

template<int SKIP, int YDIM, int NREPS>
  __global__ void __word2vecPos_exp(int nrows, int ncols, int *W, int *LB, int *UB, float *A, float *B, float lrate) {
  const int nwindow = 2*SKIP+1; 
  float aa[NREPS];
  float da[NREPS];
  __shared__ float CC[YDIM * nwindow];

  int i, j, k, tid, icol, dxy, lb, ub, iword, cword;
  float bb, db, prod, v;
  tid = threadIdx.x + blockDim.x * threadIdx.y;
  dxy = blockDim.x * blockDim.y;

  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns

    iword = nrows * W[icol];                                // Get the current word

    __syncthreads();
    
    lb = LB[icol];
    ub = UB[icol];

    if (iword >= 0) {
#pragma unroll                 
      for (j = 0; j < NREPS; j++) {                         // Iterate over blocks of elements
        if (tid + j * dxy < nrows) {                        // Get A
          aa[j] = A[tid + j * dxy + iword];
        } else {
          aa[j] = 0;
        }
      }

      for (i = lb; i <= ub; i++) {                          // Iterate across the window for A cols
        __syncthreads();
        cword = nrows * W[icol + i];                        // Get the current word
        prod = 0;
        if (cword >= 0) {
#pragma unroll                 
          for (j = 0; j < NREPS; j++) {                     // Iterate over blocks of elements
            if (tid + j * dxy < nrows) {                    // Get B col
              bb = B[tid + j * dxy + cword];
              prod += aa[j] * bb;                           // Compute the product between current A, B cols
            }
          }
#pragma unroll                 
          for (k = 1; k < 32; k = k + k) {
            prod += __shfl_down(prod, k);                   // Reduce within warp
          }  
        }
        if (threadIdx.x == 0) {
          CC[i - lb + threadIdx.y * nwindow] = prod;        // Save to SHMEM
        }
      }

      __syncthreads();
      for (j = 1; j < blockDim.y; j++) {                    // Reduce across warps
        for (i = tid; i < ub - lb; i += dxy) { 
          CC[i] += CC[i + j * nwindow];
        }
        __syncthreads();
      }

      __syncthreads();                                      //  Apply the sigmoid map
      for (i = tid; i < ub - lb; i += dxy) { 
        v = CC[i];
        if (v > 16.0f) {
          v = 1.0f;
        } else {
          v = exp(v);
          v = v / (1.0f + v);
        }
        CC[i] = lrate * (1.0f - v);                         // All pairs have label 1
      }
      
      __syncthreads();  
#pragma unroll                 
      for (j = 0; j < NREPS; j++) {
        da[j] = 0;
      }
      for (i = lb; i <= ub; i++) {                          // Iterate across the window for A cols   
        cword = nrows * W[icol + i];                        // Get the context word
        v = CC[i - lb];
        if (cword >= 0) {
#pragma unroll                 
          for (j = 0; j < NREPS; j++) {                     // Iterate over blocks of elements
            if (tid + j * dxy < nrows) {                    // Get B col
              bb = B[tid + j * dxy + cword];
              da[j] += v * bb;
              db = v * aa[j];
              atomicAdd(&B[tid + j * dxy + cword], db);
            }
          }
        }
      }

#pragma unroll                 
      for (j = 0; j < NREPS; j++) {
        if (tid + j * dxy < nrows) {                    
          atomicAdd(&A[tid + j * dxy + iword], da[j]);
        }
      }
    }
  }
}

/*
 * Combined forward-backward word2vec kernel
 */


template<int NWA, int NWB, int MAXD, int BYDIM>
  __global__ void __word2vecNeg_old(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float lrate) {
  const int NWAB = NWA*NWB;
  __shared__ float CC[NWA*NWB*BYDIM];
  float dd[MAXD];
  float prods[NWA][NWB];
  float aa, v, sum;
  int ia[NWA];
  int ib[NWB];
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int i, j, k, icol;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns
#pragma unroll
    for (i = 0; i < NWA; i++) {
      ia[i] = nrows * WA[i + icol * NWA];                   // Fill the A word matrix
#pragma unroll
      for (j = 0; j < NWB; j++) {                           // clear the products matrix
        prods[i][j] = 0;
      }
    }
#pragma unroll
    for (i = 0; i < NWB; i++) {
      ib[i] = nrows * WB[i + icol * NWB];                   // Fill the B word matrix
    }

    for (i = tid; i < nrows; i += dxy) {                    // Now iterate over the rows of this block
#pragma unroll
      for (j = 0; j < NWB ; j++) {                          // Read B
        if (ib[j] >= 0) {
          dd[j] = B[i + ib[j]];
        } else {
          dd[j] = 0;
        }
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Compute the inner products of these elements
        if (ia[j] >= 0) {
          aa = A[i + ia[j]];
#pragma unroll
          for (k = 0; k < NWB; k++) {
            prods[j][k] += aa * dd[k];
          }
        }
      }
    }                                                       // Finished the entire block

#pragma unroll
    for (i = 0; i < NWA; i++) {                             // Reduce the products within each warp
#pragma unroll
      for (j = 0; j < NWB; j++) {
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          float tmp = __shfl_down(prods[i][j], k);
          prods[i][j] += tmp;
        }
      }
    }
    __syncthreads();
    if (threadIdx.x == 0) {                                 // Save the products to SHMEM (one copy per warp)
#pragma unroll
      for (i = 0; i < NWA; i++) {
#pragma unroll
        for (j = 0; j < NWB; j++) {
          CC[j + NWB * (i + NWA * threadIdx.y)] = prods[i][j];
        }
      }
    }
    __syncthreads();
    for (i = 1; i < blockDim.y; i++) {
      __syncthreads();
      for (j = tid; j < NWAB; j += dxy) {                   // Reduce the products across warps
        CC[j] += CC[j + i * NWAB];
      } 
    } 
    __syncthreads();

    for (i = tid; i < NWA*NWB; i+= dxy) {                   // Compute logistic function on all products
      v = CC[i];
      if (v > 16.0f) {
        v = 1.0f;
      } else {
        v = exp(v);
        v = v / (1.0f + v);
      }
      CC[i] = - lrate * v;                                  // All these pairs have label 0
    }

    __syncthreads();
    for (i = tid; i < nrows; i += dxy) {
#pragma unroll
      for (j = 0; j < NWB; j++) {                           // Load B data
        if (ib[j] >= 0) {
          dd[j] = B[i + ib[j]];
        } else {
          dd[j] = 0;
        }
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Now do the product
        if (ia[j] >= 0) {
          sum = 0;
#pragma unroll
          for (k = 0; k < NWB; k++) {                       
            float xx = CC[j + k * NWA];
            sum += xx * dd[k];
          }
          atomicAdd(&A[i + ia[j]], sum);
        }
      }

#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Load A data
        if (ia[j] >= 0) {
          dd[j] = A[i + ia[j]];
        } else {
          dd[j] = 0;
        }
      }
#pragma unroll
      for (j = 0; j < NWB; j++) {                           // Now do the product
        if (ib[j] >= 0) {
          sum = 0;
#pragma unroll
          for (k = 0; k < NWA; k++) {                       
            float xx = CC[k + j * NWA];
            sum += xx * dd[k];
          }
          atomicAdd(&B[i + ib[j]], sum);
        }
      }
    } 
    __syncthreads();

  }
}

/*
 *
 * Simple forward kernel for word2vec. Computes inner products of columns from A with columns from B. 
 * The column indices are specified by two "word" matrices. The inner products are computed as an outer product
 * of the word matrices.
 * 
 *  NWA is the number of words per column in WA
 *  NWB is the number of words per column in WB
 *
 *  Columns of the output matrix C are <window> = NWA*NWB long, and contain inner products with corresponding columns of B. 
 *
 */

template<int NWA, int NWB, int BDIM>
__global__ void __word2vecFwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C) {
  const int NWAB = NWA*NWB;
  __shared__ float CC[NWA*NWB*BDIM];
  float aa;
  float bb[NWB];
  float prods[NWA][NWB];
  int wa[NWA];
  int wb[NWB];
  int tid = threadIdx.x + blockDim.x * threadIdx.y;
  int dxy = blockDim.x * blockDim.y;
  int i, j, k, icol;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

  for (icol = istart; icol < iend; icol++) {                // Iterate over columns
#pragma unroll
    for (i = 0; i < NWA; i++) {
      wa[i] = nrows * WA[i + icol * NWA];                   // Fill the A word matrix
#pragma unroll
      for (j = 0; j < NWB; j++) {                           // clear the products matrix
        prods[i][j] = 0;
      }
    }
#pragma unroll
    for (i = 0; i < NWB; i++) {
      wb[i] = WB[i + icol * NWB];                           // Fill the B word matrix
    }

    for (i = tid; i < nrows; i += dxy) {                    // Now iterate over the rows of this block
#pragma unroll
      for (j = 0; j < NWB ; j++) {                          // Read B
        bb[j] = B[i + wb[j] * nrows];
      }
#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Computes the products of these elements
        aa = A[i + wa[j] * nrows];
#pragma unroll
        for (k = 0; k < NWB; k++) {
          prods[j][k] += aa * bb[k];
        }
      }
    }                                                       // Finished the entire block

#pragma unroll
    for (i = 0; i < NWA; i++) {                             // Reduce the products within each warp
#pragma unroll
      for (j = 0; j < NWB; j++) {
#pragma unroll
        for (k = 1; k < 32; k = k+k) {
          float tmp = __shfl_down(prods[i][j], k);
          prods[i][j] += tmp;
        }
      }
    }

    __syncthreads();
    if (threadIdx.x == 0) {                                 // Save the products to SHMEM (one copy per warp)
#pragma unroll
      for (i = 0; i < NWA; i++) {
#pragma unroll
        for (j = 0; j < NWB; j++) {
          CC[j + NWB * (i + NWA * threadIdx.y)] = prods[i][j];
        }
      }
    }

    __syncthreads();
    for (i = 1; i < blockDim.y; i++) {
      __syncthreads();
#pragma unroll
      for (j = tid; j < NWAB; j += dxy) {                   // Reduce the products across warps
        CC[j] += CC[j + i * NWAB];
      } 
    } 
    __syncthreads();
    for (i = tid; i < NWAB; i += dxy) {                     // Save to main memory
      C[i + icol * NWAB] = CC[i];  
        //atomicAdd(&C[i + icol * NWAB], CC[i]); 
    }
    __syncthreads();
  }
}

/*
 *
 * Simple backward kernel for word2vec. 
 * Computes the gradient for A given B or vice-versa, and does an SGD update.
 * 
 *  NWA is the number of words per column in WA
 *  NWB is the number of words per column in WB
 *
 */


template<int NWA, int NWB, int MAXDIM>
  __global__ void __word2vecBwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C, float lrate) {
  const int NWAB = NWA * NWB;
  float dd[MAXDIM];
  int wa[NWA];
  int wb[NWB];
  __shared__ float cc[NWA*NWB];
  int tid = threadIdx.x;
  int fid = threadIdx.x + blockDim.x * threadIdx.y; 
  int dxy = blockDim.x * blockDim.y;
  int icol, i, j, k;
  float sum;
  int istart = (int)((1L * blockIdx.x * ncols) / gridDim.x);
  int iend = (int)((1L * (blockIdx.x+1) * ncols) / gridDim.x);

  for (icol = istart; icol < iend; icol++) {                // iterate in columns
#pragma unroll
    for (j = 0; j < NWA; j++) {
      wa[j] = WA[j + icol * NWA];                           // Load the A word matrix
    }
    __syncthreads();
#pragma unroll 
    for (j = 0; j < NWB; j++) {
      wb[j] = WB[j + icol * NWB];                           // Load the B word matrix
    }
    for (i = fid; i < NWAB; i += dxy) {
      cc[i] = C[i + icol * NWAB];
    }
    __syncthreads();
    for (i = tid; i < nrows; i += dxy) {
#pragma unroll
      for (j = 0; j < NWB; j++) {                           // Load the data
        dd[j] = B[i + wb[j] * nrows];
      }

#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Now do the product
        sum = 0;
#pragma unroll
        for (k = 0; k < NWB; k++) {                       
          float xx =  cc[j + k * NWA];
          sum += xx * dd[k];
        }
        atomicAdd(&A[i + wa[j] * nrows], sum * lrate);
      }

#pragma unroll
      for (j = 0; j < NWA; j++) {                           // Load the data
        dd[j] = A[i + wa[j] * nrows];
      }

#pragma unroll
      for (j = 0; j < NWB; j++) {                           // Now do the product
        sum = 0;
#pragma unroll
        for (k = 0; k < NWA; k++) {                       
          float xx =  cc[k + j * NWA];
          sum += xx * dd[k];
        }
        atomicAdd(&B[i + wb[j] * nrows], sum * lrate);
      }
    }
  }
}



#else

template<int SKIP, int BYDIM, int NREPS>
  __global__ void __word2vecPos(int nrows, int ncols, int *W, int *LB, int *UB, float *A, float *B, float lrate, float vexp) {}

template<int NWA, int NWB, int BYDIM>
  __global__ void __word2vecNeg(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float lrate, float vexp) {}

template<int NWA, int NWB, int BYDIM>
  __global__ void __word2vecNegFilt(int nrows, int ncols, int nwords, int *WA, int *WB, float *A, float *B, float lrate, float vexp) {}

template<int SKIP, int BYDIM, int NREPS>
  __global__ void __word2vecEvalPos(int nrows, int ncols, int *W, int *LB, int *UB, float *A, float *B, float *Retval) {}

template<int NWA, int NWB, int BYDIM>
  __global__ void __word2vecEvalNeg(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *Retval) {}

template<int NWA, int NWB, int BDIM>
__global__ void __word2vecFwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C) {}

template<int NWA, int NWB, int MAXDIM>
  __global__ void __word2vecBwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C, float lrate) {}

#endif

int word2vecPos(int nrows, int ncols, int skip, int *W, int *LB, int *UB, float *A, float *B, float lrate, float vexp) {
  dim3 threads(32, CDIM, 1);
  int nblocks = min(64, ncols);
  if (nrows <= 320) {
    switch(skip) {
    case 5 : __word2vecPos<5, CDIM, 10/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    case 3 : __word2vecPos<3, CDIM, 10/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    case 2 : __word2vecPos<2, CDIM, 10/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    default : printf("word2vecPos unsupport size %d\n", skip); return 1;
    }
  } else if (nrows <= 640) {
    switch(skip) {
    case 5 : __word2vecPos<5, CDIM, 20/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    case 3 : __word2vecPos<3, CDIM, 20/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    case 2 : __word2vecPos<2, CDIM, 20/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    default : printf("word2vecPos unsupport size %d\n", skip); return 1;
    }
  } else if (nrows <= 1280) {
    switch(skip) {
    case 5 : __word2vecPos<5, CDIM, 40/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    case 3 : __word2vecPos<3, CDIM, 40/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    case 2 : __word2vecPos<2, CDIM, 40/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, lrate, vexp); break;
    default : printf("word2vecPos unsupport size %d\n", skip); return 1;
    }
  } else {
    printf("word2vecPos too many rows %d\n", nrows);
    return 1;
  }        
  hipStreamSynchronize(SYNC_STREAM);
  int err = hipGetLastError();
  return err;
}


int word2vecNeg(int nrows, int ncols, int nwa, int nwb, int *WA, int *WB, float *A, float *B, float lrate, float vexp) {
  int which = nwa*10000 + nwb;
  int nblocks = min(2048, 2 + (ncols - 1));
  if (nrows <= 320) {
    const int bydim = 2;
    dim3 threads(32, bydim, 1);
    switch (which) {
    case  50001: __word2vecNeg<5,1,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate, vexp); break;
    case  50005: __word2vecNeg<5,5,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate, vexp); break;
    case 100005: __word2vecNeg<10,5,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate, vexp); break;
    case  50010: __word2vecNeg<5,10,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate, vexp); break;
      //  case 150010: __word2vecNeg<15,10,15><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate); break;
    default : printf("word2vec unsupport size combination %d %d\n", nwa, nwb); return 1;
    }
  } else {
    const int bydim = 5;
    dim3 threads(32, bydim, 1);
    switch (which) {
    case  50001: __word2vecNeg<5,1,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate, vexp); break;
    case  50005: __word2vecNeg<5,5,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate, vexp); break;
    case 100005: __word2vecNeg<10,5,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate, vexp); break;
    case  50010: __word2vecNeg<5,10,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate, vexp); break;
      //  case 150010: __word2vecNeg<15,10,15><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, lrate); break;
    default : printf("word2vec unsupport size combination %d %d\n", nwa, nwb); return 1;
    }
  }
  hipStreamSynchronize(SYNC_STREAM);
  int err = hipGetLastError();
  return err;
}

int word2vecNegFilt(int nrows, int ncols, int nwords, int nwa, int nwb, int *WA, int *WB, float *A, float *B, float lrate, float vexp) {
  int which = nwa*10000 + nwb;
  int nblocks = min(2048, 2 + (ncols - 1));
  if (nrows <= 320) {
    const int bydim = 2;
    dim3 threads(32, bydim, 1);
    switch (which) {
    case  50001: __word2vecNegFilt<5,1,bydim><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate, vexp); break;
    case  50005: __word2vecNegFilt<5,5,bydim><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate, vexp); break;
    case 100005: __word2vecNegFilt<10,5,bydim><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate, vexp); break;
    case  50010: __word2vecNegFilt<5,10,bydim><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate, vexp); break;
      //  case 150010: __word2vecNegFilt<15,10,15><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate); break;
    default : printf("word2vec unsupport size combination %d %d\n", nwa, nwb); return 1;
    }
  } else {
    const int bydim = 5;
    dim3 threads(32, bydim, 1);
    switch (which) {
    case  50001: __word2vecNegFilt<5,1,bydim><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate, vexp); break;
    case  50005: __word2vecNegFilt<5,5,bydim><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate, vexp); break;
    case 100005: __word2vecNegFilt<10,5,bydim><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate, vexp); break;
    case  50010: __word2vecNegFilt<5,10,bydim><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate, vexp); break;
      //  case 150010: __word2vecNegFilt<15,10,15><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, lrate); break;
    default : printf("word2vec unsupport size combination %d %d\n", nwa, nwb); return 1;
    }
  }
  hipStreamSynchronize(SYNC_STREAM);
  int err = hipGetLastError();
  return err;
}

int word2vecEvalPos(int nrows, int ncols, int skip, int *W, int *LB, int *UB, float *A, float *B, float *Retval) {
  dim3 threads(32, CDIM, 1);
  int nblocks = min(64, ncols);
  if (nrows <= 320) {
    switch(skip) {
    case 5 : __word2vecEvalPos<5, CDIM, 10/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    case 3 : __word2vecEvalPos<3, CDIM, 10/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    case 2 : __word2vecEvalPos<2, CDIM, 10/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    default : printf("word2vecEvalPos unsupport size %d\n", skip); return 1;
    }
  } else if (nrows <= 640) {
    switch(skip) {
    case 5 : __word2vecEvalPos<5, CDIM, 20/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    case 3 : __word2vecEvalPos<3, CDIM, 20/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    case 2 : __word2vecEvalPos<2, CDIM, 20/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    default : printf("word2vecEvalPos unsupport size %d\n", skip); return 1;
    }
  } else if (nrows <= 1280) {
    switch(skip) {
    case 5 : __word2vecEvalPos<5, CDIM, 40/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    case 3 : __word2vecEvalPos<3, CDIM, 40/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    case 2 : __word2vecEvalPos<2, CDIM, 40/CDIM><<<nblocks,threads>>>(nrows, ncols, W, LB, UB, A, B, Retval); break;
    default : printf("word2vecEvalPos unsupport size %d\n", skip); return 1;
    }
  } else {
    printf("word2vecEvalPos nrows too large %d\n", nrows);
    return 1;
  }
  hipStreamSynchronize(SYNC_STREAM);
  int err = hipGetLastError();
  return err;
}


int word2vecEvalNeg(int nrows, int ncols, int nwa, int nwb, int *WA, int *WB, float *A, float *B, float *Retval) {
  int which = nwa*10000 + nwb;
  int nblocks = min(2048, 2 + (ncols - 1));
  if (nrows <= 320) {
    const int bydim = 2;
    dim3 threads(32, bydim, 1);
    switch (which) {
    case 50001: __word2vecEvalNeg<5,1,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, Retval); break;
    case 50005: __word2vecEvalNeg<5,5,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, Retval); break;
    case 100005: __word2vecEvalNeg<10,5,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, Retval); break;
    case 50010: __word2vecEvalNeg<5,10,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, Retval); break;
    //  case 150010: __word2vecEvalNeg<15,10,15><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, Retval); break;
    default : printf("word2vecEvalNeg unsupport size combination %d %d\n", nwa, nwb); return 1;
    }
  } else {
    const int bydim = 5;
    dim3 threads(32, bydim, 1);
    switch (which) {
    case 50001: __word2vecEvalNeg<5,1,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, Retval); break;
    case 50005: __word2vecEvalNeg<5,5,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, Retval); break;
    case 100005: __word2vecEvalNeg<10,5,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, Retval); break;
    case 50010: __word2vecEvalNeg<5,10,bydim><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, Retval); break;
    //  case 150010: __word2vecEvalNeg<15,10,15><<<nblocks,threads>>>(nrows, ncols, nwords, WA, WB, A, B, Retval); break;
    default : printf("word2vecEvalNeg unsupport size combination %d %d\n", nwa, nwb); return 1;
    }
  }
  hipStreamSynchronize(SYNC_STREAM);
  int err = hipGetLastError();
  return err;
}

int word2vecFwd(int nrows, int ncols, int nwa, int nwb, int *WA, int *WB, float *A, float *B, float *C) {
  dim3 threads(32, BYDIMF, 1);
  int nblocks = min(4096, 2 + (ncols - 1));
  int which = nwa*10000 + nwb;
  switch (which) {
  case 50001: __word2vecFwd<5,1,BYDIMF><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C); break;
  case 50005: __word2vecFwd<5,5,BYDIMF><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C); break;
  case 100005: __word2vecFwd<10,5,BYDIMF><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C); break;
  default : printf("word2vecFwd unsupport size combination %d %d\n", nwa, nwb); return 1;
  }
  hipStreamSynchronize(SYNC_STREAM);
  int err = hipGetLastError();
  return err;
  }

int word2vecBwd(int nrows, int ncols, int nwa, int nwb, int *WA, int *WB, float *A, float *B, float *C, float lrate) {
  dim3 threads(32*BYDIMB, 1, 1);
  int nblocks = min(2048, 2 + (ncols - 1));
  int which = nwa*10000 + nwb;
  switch (which) {
  case 50001: __word2vecBwd<5,1,5><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C, lrate); break;
  case 50005: __word2vecBwd<5,5,5><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C, lrate); break;
  case 100005: __word2vecBwd<10,5,10><<<nblocks,threads>>>(nrows, ncols, WA, WB, A, B, C, lrate); break;
  default : printf("word2vecBwd unsupport size combination %d %d\n", nwa, nwb); return 1;
  }
  hipStreamSynchronize(SYNC_STREAM);
  int err = hipGetLastError();
  return err;
}
 
